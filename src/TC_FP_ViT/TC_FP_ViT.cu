#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <mma.h>

#include "TC_FP_Functions.cuh"

#define warp_size 32
#define block_size1 32
#define block_size2 24

#define TOTAL_WIDTH 784
#define TC_WIDTH 720
#define CC_WIDTH 64

void initializeRandom_int8(int8_t* array, int size) {
    for (int i = 0; i < size; ++i) {
        array[i] = rand() % 11;
    }
}

void initializeRandom_int(int* array, int size) {
    for (int i = 0; i < size; ++i) {
        array[i] = rand() % 11;
    }
}

void initializeRandom_float(float* array, int size) {
    for (int i = 0; i < size; ++i) {
        array[i] = static_cast<float>(rand()) / RAND_MAX; // 0부터 1까지의 랜덤값
    }
}

void rearrange_int8(int8_t *input, int8_t *output, int output_rows, int output_cols) {
    for (int row = 0; row < output_rows; ++row) {
        for (int col = 0; col < output_cols; ++col) {
            int idx = row * output_cols + col;
            if (row < output_rows && col < output_cols) {
                int out_row = idx / output_cols;
                int out_col = idx % output_cols;
                int in_idx = (out_row * 14 + out_col / 48) * 224 + out_col % 48;

                output[idx] = input[in_idx];
            }
        }
    }
}

void rearrange_int(int *input, int *output, int output_rows, int output_cols) {
    for (int row = 0; row < output_rows; ++row) {
        for (int col = 0; col < output_cols; ++col) {
            int idx = row * output_cols + col;
            if (row < output_rows && col < output_cols) {
                int out_row = idx / output_cols;
                int out_col = idx % output_cols;
                int in_idx = (out_row * 14 + out_col / 48) * 224 + out_col % 48;

                output[idx] = input[in_idx];
            }
        }
    }
}

int main(){

    ///// Initalizing Input Data, Weight, Bias, Gamma, Beta /////
    /* Rearange 1 */
    int8_t *Rearrange_TC_input = new int8_t[224 * 224 * 3 * TC_WIDTH / (TC_WIDTH + CC_WIDTH)];
    initializeRandom_int8(Rearrange_TC_input, 224 * 224 * 3 * TC_WIDTH / (TC_WIDTH + CC_WIDTH));
    float *Rearrange_CC_input = new float[224 * 224 * 3 * CC_WIDTH / (TC_WIDTH + CC_WIDTH)];
    initializeRandom_float(Rearrange_CC_input, 224 * 224 * 3 * CC_WIDTH / (TC_WIDTH + CC_WIDTH));

    /* Measuring Preprocessng Time*/
    // clock_t Pre_Processing_Start, Pre_Processing_End;
    // double Pre_Processing_Time;

    // Pre_Processing_Start = clock();

    int8_t *Rearrange1_TC_output_GPU;
    hipMalloc(&Rearrange1_TC_output_GPU, (TC_WIDTH*192) * sizeof(int8_t));
    hipMemcpy(Rearrange1_TC_output_GPU, Rearrange_TC_input, (TC_WIDTH*192) * sizeof(int8_t), hipMemcpyHostToDevice);
    float *Rearrange1_CC_output_GPU;
    hipMalloc(&Rearrange1_CC_output_GPU, (CC_WIDTH*192) * sizeof(float));
    hipMemcpy(Rearrange1_CC_output_GPU, Rearrange_CC_input, (CC_WIDTH*192) * sizeof(float), hipMemcpyHostToDevice);

    /* Layer Normalization 2 */
    int8_t *Norm2_TC_gamma_CPU = new int8_t[192];
    initializeRandom_int8(Norm2_TC_gamma_CPU, 192);
    float *Norm2_CC_gamma_CPU = new float[192];
    initializeRandom_float(Norm2_CC_gamma_CPU, 192);

    int8_t *Norm2_TC_gamma_GPU;
    hipMalloc(&Norm2_TC_gamma_GPU, 192 * sizeof(int8_t));
    hipMemcpy(Norm2_TC_gamma_GPU, Norm2_TC_gamma_CPU, 192 * sizeof(int8_t), hipMemcpyHostToDevice);
    float *Norm2_CC_gamma_GPU;
    hipMalloc(&Norm2_CC_gamma_GPU, 192 * sizeof(float));
    hipMemcpy(Norm2_CC_gamma_GPU, Norm2_CC_gamma_CPU, 192 * sizeof(float), hipMemcpyHostToDevice);

    int8_t *Norm2_TC_beta_CPU = new int8_t[192];
    initializeRandom_int8(Norm2_TC_beta_CPU, 192);
    int8_t *Norm2_TC_beta_GPU;
    hipMalloc(&Norm2_TC_beta_GPU, 192 * sizeof(int8_t));
    hipMemcpy(Norm2_TC_beta_GPU, Norm2_TC_beta_CPU, 192 * sizeof(int8_t), hipMemcpyHostToDevice);
    float *Norm2_CC_beta_CPU = new float[192];
    initializeRandom_float(Norm2_CC_beta_CPU, 192);
    float *Norm2_CC_beta_GPU;
    hipMalloc(&Norm2_CC_beta_GPU, 192 * sizeof(float));
    hipMemcpy(Norm2_CC_beta_GPU, Norm2_CC_beta_CPU, 192 * sizeof(float), hipMemcpyHostToDevice);

    int8_t *Norm2_TC_output;
    hipMalloc(&Norm2_TC_output, (TC_WIDTH*192) * sizeof(int8_t));
    float *Norm2_CC_output;
    hipMalloc(&Norm2_CC_output, (CC_WIDTH*192) * sizeof(float));

    /* Linear 3 */
    // TC Parameters
    int8_t *Linear3_TC_weight_CPU = new int8_t[192 * 768];
    initializeRandom_int8(Linear3_TC_weight_CPU, 192 * 768);
    int8_t *Linear3_TC_weight_GPU;
    hipMalloc(&Linear3_TC_weight_GPU, (192 * 768) * sizeof(int8_t));
    hipMemcpy(Linear3_TC_weight_GPU, Linear3_TC_weight_CPU, (192 * 768) * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Linear3_TC_bias_CPU = new int8_t[768];
    initializeRandom_int8(Linear3_TC_bias_CPU, 768);
    int8_t *Linear3_TC_bias_GPU;
    hipMalloc(&Linear3_TC_bias_GPU, 768 * sizeof(int8_t));
    hipMemcpy(Linear3_TC_bias_GPU, Linear3_TC_bias_CPU, 768 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Linear3_TC_output;
    hipMalloc(&Linear3_TC_output, (TC_WIDTH*768) * sizeof(int8_t));
    int8_t *Linear3_1_TC_output;
    hipMalloc(&Linear3_1_TC_output, (TC_WIDTH*768) * sizeof(int8_t));

    // CC Parameters
    float *Linear3_CC_weight_CPU = new float[192 * 768];
    initializeRandom_float(Linear3_CC_weight_CPU, 192 * 768);
    float *Linear3_CC_weight_GPU;
    hipMalloc(&Linear3_CC_weight_GPU, (192 * 768) * sizeof(float));
    hipMemcpy(Linear3_CC_weight_GPU, Linear3_CC_weight_CPU, (192 * 768) * sizeof(float), hipMemcpyHostToDevice);

    float *Linear3_CC_bias_CPU = new float[768];
    initializeRandom_float(Linear3_CC_bias_CPU, 768);
    float *Linear3_CC_bias_GPU;
    hipMalloc(&Linear3_CC_bias_GPU, 768 * sizeof(float));
    hipMemcpy(Linear3_CC_bias_GPU, Linear3_CC_bias_CPU, 768 * sizeof(float), hipMemcpyHostToDevice);

    float *Linear3_CC_output;
    hipMalloc(&Linear3_CC_output, (CC_WIDTH*768) * sizeof(float));
    float *Linear3_1_CC_output;
    hipMalloc(&Linear3_1_CC_output, (CC_WIDTH*768) * sizeof(float));

    /* Layer Normalization 4 */
    // TC Parameters
    int8_t *Norm4_TC_gamma_CPU = new int8_t[768];
    initializeRandom_int8(Norm4_TC_gamma_CPU, 768);
    int8_t *Norm4_TC_gamma_GPU;
    hipMalloc(&Norm4_TC_gamma_GPU, 768 * sizeof(int8_t));
    hipMemcpy(Norm4_TC_gamma_GPU, Norm4_TC_gamma_CPU, 768 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Norm4_TC_beta_CPU = new int8_t[768];
    initializeRandom_int8(Norm4_TC_beta_CPU, 768);
    int8_t *Norm4_TC_beta_GPU;
    hipMalloc(&Norm4_TC_beta_GPU, 768 * sizeof(int8_t));
    hipMemcpy(Norm4_TC_beta_GPU, Norm4_TC_beta_CPU, 768 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Norm4_TC_output;
    hipMalloc(&Norm4_TC_output, (TC_WIDTH*768) * sizeof(int8_t));

    // CC Parameters
    float *Norm4_CC_gamma_CPU = new float[768];
    initializeRandom_float(Norm4_CC_gamma_CPU, 768);
    float *Norm4_CC_gamma_GPU;
    hipMalloc(&Norm4_CC_gamma_GPU, 768 * sizeof(float));
    hipMemcpy(Norm4_CC_gamma_GPU, Norm4_CC_gamma_CPU, 768 * sizeof(float), hipMemcpyHostToDevice);

    float *Norm4_CC_beta_CPU = new float[768];
    initializeRandom_float(Norm4_CC_beta_CPU, 768);
    float *Norm4_CC_beta_GPU;
    hipMalloc(&Norm4_CC_beta_GPU, 768 * sizeof(float));
    hipMemcpy(Norm4_CC_beta_GPU, Norm4_CC_beta_CPU, 768 * sizeof(float), hipMemcpyHostToDevice);

    float *Norm4_CC_output;
    hipMalloc(&Norm4_CC_output, (CC_WIDTH*768) * sizeof(float));

    /* Dropout 5 */
    // TC Parameters
    int8_t *Drop5_TC_output;
    hipMalloc(&Drop5_TC_output, (TC_WIDTH*768) * sizeof(int8_t));

    // CC Parameters
    float *Drop5_CC_output;
    hipMalloc(&Drop5_CC_output, (CC_WIDTH*768) * sizeof(float));

    const float dropout_prob = 0.5f;

    /* Layer Normalization 6 */
    // TC Parameters
    int8_t *Norm6_TC_gamma_CPU = new int8_t[768];
    initializeRandom_int8(Norm6_TC_gamma_CPU, 768);
    int8_t *Norm6_TC_gamma_GPU;
    hipMalloc(&Norm6_TC_gamma_GPU, 768 * sizeof(int8_t));
    hipMemcpy(Norm6_TC_gamma_GPU, Norm6_TC_gamma_CPU, 768 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Norm6_TC_beta_CPU = new int8_t[768];
    initializeRandom_int8(Norm6_TC_beta_CPU, 768);
    int8_t *Norm6_TC_beta_GPU;
    hipMalloc(&Norm6_TC_beta_GPU, 768 * sizeof(int8_t));
    hipMemcpy(Norm6_TC_beta_GPU, Norm6_TC_beta_CPU, 768 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Norm6_TC_output;
    hipMalloc(&Norm6_TC_output, (TC_WIDTH*768) * sizeof(int8_t));

    // CC Parameters
    float *Norm6_CC_gamma_CPU = new float[768];
    initializeRandom_float(Norm6_CC_gamma_CPU, 768);
    float *Norm6_CC_gamma_GPU;
    hipMalloc(&Norm6_CC_gamma_GPU, 768 * sizeof(float));
    hipMemcpy(Norm6_CC_gamma_GPU, Norm6_CC_gamma_CPU, 768 * sizeof(float), hipMemcpyHostToDevice);

    float *Norm6_CC_beta_CPU = new float[768];
    initializeRandom_float(Norm6_CC_beta_CPU, 768);
    float *Norm6_CC_beta_GPU;
    hipMalloc(&Norm6_CC_beta_GPU, 768 * sizeof(float));
    hipMemcpy(Norm6_CC_beta_GPU, Norm6_CC_beta_CPU, 768 * sizeof(float), hipMemcpyHostToDevice);

    float *Norm6_CC_output;
    hipMalloc(&Norm6_CC_output, (CC_WIDTH*768) * sizeof(float));

    //// Iteration Start
    int num_iteration = 12;

    /* Linear 7 */
    // TC Parameters
    int8_t *Linear7_TC_weight_CPU[num_iteration];
    int8_t *Linear7_TC_weight_GPU[num_iteration];
    int8_t *Linear7_TC_bias_CPU[num_iteration];
    int8_t *Linear7_TC_bias_GPU[num_iteration];
    int8_t *Linear7_TC_output[num_iteration];
    int8_t *Linear7_1_TC_output[num_iteration];

    // CC Parameters
    float *Linear7_CC_weight_CPU[num_iteration];
    float *Linear7_CC_weight_GPU[num_iteration];
    float *Linear7_CC_bias_CPU[num_iteration];
    float *Linear7_CC_bias_GPU[num_iteration];
    float *Linear7_CC_output[num_iteration];
    float *Linear7_1_CC_output[num_iteration];

    /* Softmax 8 */
    // TC Parameters
    int8_t *Soft8_TC_output[num_iteration];

    // CC Parameters
    float *Soft8_CC_output[num_iteration];

    /* Dropout 9 */
    // TC Parametsers
    int8_t *Drop9_TC_output[num_iteration];

    // CC Parametsers
    float *Drop9_CC_output[num_iteration];

    /* Linear 10 */
    // TC Parameters
    int8_t *Linear10_TC_weight_CPU[num_iteration];
    int8_t *Linear10_TC_weight_GPU[num_iteration];
    int8_t *Linear10_TC_bias_CPU[num_iteration];
    int8_t *Linear10_TC_bias_GPU[num_iteration];
    int8_t *Linear10_TC_output[num_iteration];
    int8_t *Linear10_1_TC_output[num_iteration];

    // CC Parameters
    float *Linear10_CC_weight_CPU[num_iteration];
    float *Linear10_CC_weight_GPU[num_iteration];
    float *Linear10_CC_bias_CPU[num_iteration];
    float *Linear10_CC_bias_GPU[num_iteration];
    float *Linear10_CC_output[num_iteration];
    float *Linear10_1_CC_output[num_iteration];

    /* Dropout 11 */
    // TC Parameters
    int8_t *Drop11_TC_output[num_iteration];

    // CC Parameters
    float *Drop11_CC_output[num_iteration];

    /* Layer Normalization 13 */
    // TC Parameters
    int8_t *Norm13_TC_gamma_CPU[num_iteration];
    int8_t *Norm13_TC_gamma_GPU[num_iteration];
    int8_t *Norm13_TC_beta_CPU[num_iteration];
    int8_t *Norm13_TC_beta_GPU[num_iteration];
    int8_t *Norm13_TC_output[num_iteration];

    // CC Parameters
    float *Norm13_CC_gamma_CPU[num_iteration];
    float *Norm13_CC_gamma_GPU[num_iteration];
    float *Norm13_CC_beta_CPU[num_iteration];
    float *Norm13_CC_beta_GPU[num_iteration];
    float *Norm13_CC_output[num_iteration];

    /* Linear 14 */
    // TC Parameters
    int8_t *Linear14_TC_weight_CPU[num_iteration];
    int8_t *Linear14_TC_weight_GPU[num_iteration];
    int8_t *Linear14_TC_bias_CPU[num_iteration];
    int8_t *Linear14_TC_bias_GPU[num_iteration];
    int8_t *Linear14_TC_output[num_iteration];
    int8_t *Linear14_1_TC_output[num_iteration];

    // CC Parameters
    float *Linear14_CC_weight_CPU[num_iteration];
    float *Linear14_CC_weight_GPU[num_iteration];
    float *Linear14_CC_bias_CPU[num_iteration];
    float *Linear14_CC_bias_GPU[num_iteration];
    float *Linear14_CC_output[num_iteration];
    float *Linear14_1_CC_output[num_iteration];

    /* Gelu 15 */
    // TC Parameters
    int8_t *Gelu15_TC_output[num_iteration];

    // CC Parameters
    float *Gelu15_CC_output[num_iteration];

    /* Dropout 16 */
    // TC Parameters
    int8_t *Drop16_TC_output[num_iteration];

    // CC Parameters
    float *Drop16_CC_output[num_iteration];

    /* Linear 17 */
    // TC Parameters
    int8_t *Linear17_TC_weight_CPU[num_iteration];
    int8_t *Linear17_TC_weight_GPU[num_iteration];
    int8_t *Linear17_TC_bias_CPU[num_iteration];
    int8_t *Linear17_TC_bias_GPU[num_iteration];
    int8_t *Linear17_TC_output[num_iteration];
    int8_t *Linear17_1_TC_output[num_iteration];

    // CC Parameters
    float *Linear17_CC_weight_CPU[num_iteration];
    float *Linear17_CC_weight_GPU[num_iteration];
    float *Linear17_CC_bias_CPU[num_iteration];
    float *Linear17_CC_bias_GPU[num_iteration];
    float *Linear17_CC_output[num_iteration];
    float *Linear17_1_CC_output[num_iteration];

    /* Dropout 18 */
    // TC Parameters
    int8_t *Drop18_TC_output[num_iteration];

    // CC Parameters
    float *Drop18_CC_output[num_iteration];

    /* Layer Normalization 20 */
    // TC Parameters
    int8_t *Norm20_TC_gamma_CPU[num_iteration];
    int8_t *Norm20_TC_gamma_GPU[num_iteration];
    int8_t *Norm20_TC_beta_CPU[num_iteration];
    int8_t *Norm20_TC_beta_GPU[num_iteration];
    int8_t *Norm20_TC_output[num_iteration];

    // CC Parameters
    float *Norm20_CC_gamma_CPU[num_iteration];
    float *Norm20_CC_gamma_GPU[num_iteration];
    float *Norm20_CC_beta_CPU[num_iteration];
    float *Norm20_CC_beta_GPU[num_iteration];
    float *Norm20_CC_output[num_iteration];
    
    for(int i = 0; i < num_iteration; ++i){
        /* Linear 7 */
        // TC Parameters
        Linear7_TC_weight_CPU[i] = new int8_t[768 * 2304];
        initializeRandom_int8(Linear7_TC_weight_CPU[i], 768 * 2304);
        hipMalloc(&Linear7_TC_weight_GPU[i], (768 * 2304) * sizeof(int8_t));
        hipMemcpy(Linear7_TC_weight_GPU[i], Linear7_TC_weight_CPU[i], (768 * 2304) * sizeof(int8_t), hipMemcpyHostToDevice);
        Linear7_TC_bias_CPU[i] = new int8_t[2304];
        initializeRandom_int8(Linear7_TC_bias_CPU[i], 2304);   
        hipMalloc(&Linear7_TC_bias_GPU[i], 2304 * sizeof(int8_t));
        hipMemcpy(Linear7_TC_bias_GPU[i], Linear7_TC_bias_CPU[i], 2304 * sizeof(int8_t), hipMemcpyHostToDevice);
        hipMalloc(&Linear7_TC_output[i], (TC_WIDTH*2304) * sizeof(int8_t));
        hipMalloc(&Linear7_1_TC_output[i], (TC_WIDTH*2304) * sizeof(int8_t));

        // CC Parameters
        Linear7_CC_weight_CPU[i] = new float[768 * 2304];
        initializeRandom_float(Linear7_CC_weight_CPU[i], 768 * 2304);
        hipMalloc(&Linear7_CC_weight_GPU[i], (768 * 2304) * sizeof(float));
        hipMemcpy(Linear7_CC_weight_GPU[i], Linear7_CC_weight_CPU[i], (768 * 2304) * sizeof(float), hipMemcpyHostToDevice);
        Linear7_CC_bias_CPU[i] = new float[2304];
        initializeRandom_float(Linear7_CC_bias_CPU[i], 2304);   
        hipMalloc(&Linear7_CC_bias_GPU[i], 2304 * sizeof(float));
        hipMemcpy(Linear7_CC_bias_GPU[i], Linear7_CC_bias_CPU[i], 2304 * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&Linear7_CC_output[i], (CC_WIDTH*2304) * sizeof(float));
        hipMalloc(&Linear7_1_CC_output[i], (CC_WIDTH*2304) * sizeof(float));

        /* Softmax 8 */
        // TC Parameters
        hipMalloc(&Soft8_TC_output[i], (TC_WIDTH*2304) * sizeof(int8_t));

        // CC Parameters
        hipMalloc(&Soft8_CC_output[i], (CC_WIDTH*2304) * sizeof(float));

        /* Dropout 9 */
        // TC Parameters
        hipMalloc(&Drop9_TC_output[i], (TC_WIDTH*768) * sizeof(int8_t));

        // CC Parameters
        hipMalloc(&Drop9_CC_output[i], (CC_WIDTH*768) * sizeof(float));

        /* Linear 10 */
        // TC Parameters
        Linear10_TC_weight_CPU[i] = new int8_t[768 * 768];
        initializeRandom_int8(Linear10_TC_weight_CPU[i], 768 * 768);
        hipMalloc(&Linear10_TC_weight_GPU[i], (768 * 768) * sizeof(int8_t));
        hipMemcpy(Linear10_TC_weight_GPU[i], Linear10_TC_weight_CPU[i], (768 * 768) * sizeof(int8_t), hipMemcpyHostToDevice);
        Linear10_TC_bias_CPU[i] = new int8_t[768];
        initializeRandom_int8(Linear10_TC_bias_CPU[i], 768);
        hipMalloc(&Linear10_TC_bias_GPU[i], 768 * sizeof(int8_t));
        hipMemcpy(Linear10_TC_bias_GPU[i], Linear10_TC_bias_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);
        hipMalloc(&Linear10_TC_output[i], (TC_WIDTH*768) * sizeof(int8_t));
        hipMalloc(&Linear10_1_TC_output[i], (TC_WIDTH*768) * sizeof(int8_t));

        // CC Parameters
        Linear10_CC_weight_CPU[i] = new float[768 * 768];
        initializeRandom_float(Linear10_CC_weight_CPU[i], 768 * 768);
        hipMalloc(&Linear10_CC_weight_GPU[i], (768 * 768) * sizeof(float));
        hipMemcpy(Linear10_CC_weight_GPU[i], Linear10_CC_weight_CPU[i], (768 * 768) * sizeof(float), hipMemcpyHostToDevice);
        Linear10_CC_bias_CPU[i] = new float[768];
        initializeRandom_float(Linear10_CC_bias_CPU[i], 768);
        hipMalloc(&Linear10_CC_bias_GPU[i], 768 * sizeof(float));
        hipMemcpy(Linear10_CC_bias_GPU[i], Linear10_CC_bias_CPU[i], 768 * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&Linear10_CC_output[i], (CC_WIDTH*768) * sizeof(float));
        hipMalloc(&Linear10_1_CC_output[i], (CC_WIDTH*768) * sizeof(float));

        /* Dropout 11 */
        // TC Parameters
        hipMalloc(&Drop11_TC_output[i], (TC_WIDTH*768) * sizeof(int8_t));

        // CC Parameters
        hipMalloc(&Drop11_CC_output[i], (CC_WIDTH*768) * sizeof(float));

        /* Layer Normalization 13 */
        // TC Parameters
        Norm13_TC_gamma_CPU[i] = new int8_t[768];
        initializeRandom_int8(Norm13_TC_gamma_CPU[i], 768);
        hipMalloc(&Norm13_TC_gamma_GPU[i], 768 * sizeof(int8_t));
        hipMemcpy(Norm13_TC_gamma_GPU[i], Norm13_TC_gamma_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);
        Norm13_TC_beta_CPU[i] = new int8_t[768];
        initializeRandom_int8(Norm13_TC_beta_CPU[i], 768);
        hipMalloc(&Norm13_TC_beta_GPU[i], 768 * sizeof(int8_t));
        hipMemcpy(Norm13_TC_beta_GPU[i], Norm13_TC_beta_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);
        hipMalloc(&Norm13_TC_output[i], (TC_WIDTH*768) * sizeof(int8_t));

        // CC Parameters
        Norm13_CC_gamma_CPU[i] = new float[768];
        initializeRandom_float(Norm13_CC_gamma_CPU[i], 768);
        hipMalloc(&Norm13_CC_gamma_GPU[i], 768 * sizeof(float));
        hipMemcpy(Norm13_CC_gamma_GPU[i], Norm13_CC_gamma_CPU[i], 768 * sizeof(float), hipMemcpyHostToDevice);
        Norm13_CC_beta_CPU[i] = new float[768];
        initializeRandom_float(Norm13_CC_beta_CPU[i], 768);
        hipMalloc(&Norm13_CC_beta_GPU[i], 768 * sizeof(float));
        hipMemcpy(Norm13_CC_beta_GPU[i], Norm13_CC_beta_CPU[i], 768 * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&Norm13_CC_output[i], (CC_WIDTH*768) * sizeof(float));

        /* Linear 14 */
        // TC Parameters
        Linear14_TC_weight_CPU[i] = new int8_t[768 * 3072];
        initializeRandom_int8(Linear14_TC_weight_CPU[i], 768 * 3072);
        hipMalloc(&Linear14_TC_weight_GPU[i], (768 * 3072) * sizeof(int8_t));
        hipMemcpy(Linear14_TC_weight_GPU[i], Linear14_TC_weight_CPU[i], (768 * 3072) * sizeof(int8_t), hipMemcpyHostToDevice);
        Linear14_TC_bias_CPU[i] = new int8_t[3072];
        initializeRandom_int8(Linear14_TC_bias_CPU[i], 3072);
        hipMalloc(&Linear14_TC_bias_GPU[i], 3072 * sizeof(int8_t));
        hipMemcpy(Linear14_TC_bias_GPU[i], Linear14_TC_bias_CPU[i], 3072 * sizeof(int8_t), hipMemcpyHostToDevice);
        hipMalloc(&Linear14_TC_output[i], (TC_WIDTH*3072) * sizeof(int8_t));
        hipMalloc(&Linear14_1_TC_output[i], (TC_WIDTH*3072) * sizeof(int8_t));

        // CC Parameters
        Linear14_CC_weight_CPU[i] = new float[768 * 3072];
        initializeRandom_float(Linear14_CC_weight_CPU[i], 768 * 3072);
        hipMalloc(&Linear14_CC_weight_GPU[i], (768 * 3072) * sizeof(float));
        hipMemcpy(Linear14_CC_weight_GPU[i], Linear14_CC_weight_CPU[i], (768 * 3072) * sizeof(float), hipMemcpyHostToDevice);
        Linear14_CC_bias_CPU[i] = new float[3072];
        initializeRandom_float(Linear14_CC_bias_CPU[i], 3072);
        hipMalloc(&Linear14_CC_bias_GPU[i], 3072 * sizeof(float));
        hipMemcpy(Linear14_CC_bias_GPU[i], Linear14_CC_bias_CPU[i], 3072 * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&Linear14_CC_output[i], (CC_WIDTH*3072) * sizeof(float));
        hipMalloc(&Linear14_1_CC_output[i], (CC_WIDTH*3072) * sizeof(float));

        /* Gelu 15 */
        // TC Parameters
        hipMalloc(&Gelu15_TC_output[i], (TC_WIDTH*3072) * sizeof(int8_t));

        // CC Parameters
        hipMalloc(&Gelu15_CC_output[i], (CC_WIDTH*3072) * sizeof(float));

        /* Dropout 16 */
        // TC Parameters
        hipMalloc(&Drop16_TC_output[i], (TC_WIDTH*3072) * sizeof(int8_t));

        // CC Parameters
        hipMalloc(&Drop16_CC_output[i], (CC_WIDTH*3072) * sizeof(float));

        /* Linear 17 */
        // TC Parameters
        Linear17_TC_weight_CPU[i] = new int8_t[768 * 3072];
        initializeRandom_int8(Linear17_TC_weight_CPU[i], 768 * 3072);
        hipMalloc(&Linear17_TC_weight_GPU[i], (768 * 3072) * sizeof(int8_t));
        hipMemcpy(Linear17_TC_weight_GPU[i], Linear17_TC_weight_CPU[i], (768 * 3072) * sizeof(int8_t), hipMemcpyHostToDevice);
        Linear17_TC_bias_CPU[i] = new int8_t[768];
        initializeRandom_int8(Linear17_TC_bias_CPU[i], 768);
        hipMalloc(&Linear17_TC_bias_GPU[i], 3072 * sizeof(int8_t));
        hipMemcpy(Linear17_TC_bias_GPU[i], Linear17_TC_bias_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);
        hipMalloc(&Linear17_TC_output[i], (TC_WIDTH*768) * sizeof(int8_t));
        hipMalloc(&Linear17_1_TC_output[i], (TC_WIDTH*768) * sizeof(int8_t));

        // CC Parameters
        Linear17_CC_weight_CPU[i] = new float[768 * 3072];
        initializeRandom_float(Linear17_CC_weight_CPU[i], 768 * 3072);
        hipMalloc(&Linear17_CC_weight_GPU[i], (768 * 3072) * sizeof(float));
        hipMemcpy(Linear17_CC_weight_GPU[i], Linear17_CC_weight_CPU[i], (768 * 3072) * sizeof(float), hipMemcpyHostToDevice);
        Linear17_CC_bias_CPU[i] = new float[768];
        initializeRandom_float(Linear17_CC_bias_CPU[i], 768);
        hipMalloc(&Linear17_CC_bias_GPU[i], 3072 * sizeof(float));
        hipMemcpy(Linear17_CC_bias_GPU[i], Linear17_CC_bias_CPU[i], 768 * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&Linear17_CC_output[i], (CC_WIDTH*768) * sizeof(float));
        hipMalloc(&Linear17_1_CC_output[i], (CC_WIDTH*768) * sizeof(float));

        /* Dropout 18 */
        // TC Parameters
        hipMalloc(&Drop18_TC_output[i], (TC_WIDTH*768) * sizeof(int8_t));

        // CC Parameters
        hipMalloc(&Drop18_CC_output[i], (CC_WIDTH*768) * sizeof(float));

        /* Layer Normalization 20 */
        // TC Parameters
        Norm20_TC_gamma_CPU[i] = new int8_t[768];
        initializeRandom_int8(Norm20_TC_gamma_CPU[i], 768);
        hipMalloc(&Norm20_TC_gamma_GPU[i], 768 * sizeof(int8_t));
        hipMemcpy(Norm20_TC_gamma_GPU[i], Norm20_TC_gamma_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);
        Norm20_TC_beta_CPU[i] = new int8_t[768];
        initializeRandom_int8(Norm20_TC_beta_CPU[i], 768);
        hipMalloc(&Norm20_TC_beta_GPU[i], 768 * sizeof(int8_t));
        hipMemcpy(Norm20_TC_beta_GPU[i], Norm20_TC_beta_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);
        hipMalloc(&Norm20_TC_output[i], (TC_WIDTH*768) * sizeof(int8_t));

        // CC Parameters
        Norm20_CC_gamma_CPU[i] = new float[768];
        initializeRandom_float(Norm20_CC_gamma_CPU[i], 768);
        hipMalloc(&Norm20_CC_gamma_GPU[i], 768 * sizeof(float));
        hipMemcpy(Norm20_CC_gamma_GPU[i], Norm20_CC_gamma_CPU[i], 768 * sizeof(float), hipMemcpyHostToDevice);
        Norm20_CC_beta_CPU[i] = new float[768];
        initializeRandom_float(Norm20_CC_beta_CPU[i], 768);
        hipMalloc(&Norm20_CC_beta_GPU[i], 768 * sizeof(float));
        hipMemcpy(Norm20_CC_beta_GPU[i], Norm20_CC_beta_CPU[i], 768 * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&Norm20_CC_output[i], (CC_WIDTH*768) * sizeof(float));
    }

    // Pre_Processing_End = clock();

    // Pre_Processing_Time = ((double) (Pre_Processing_End - Pre_Processing_Start)) / CLOCKS_PER_SEC;

    // printf("Pre_Processing_Time: %f s\n", Pre_Processing_Time);

    ///// Starting Computation /////

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    dim3 dimBlock(block_size1, block_size2);

    /* Layer Normalization 2 */
    dim3 dimGrid2(ceil(192/block_size1), ceil(TOTAL_WIDTH/block_size2));
    TC_FP_Normalization<<<dimGrid2,dimBlock>>>(Rearrange1_TC_output_GPU, Norm2_TC_output, Norm2_TC_gamma_GPU, Norm2_TC_beta_GPU,
                                                Rearrange1_CC_output_GPU, Norm2_CC_output, Norm2_CC_gamma_GPU, Norm2_CC_beta_GPU,
                                                768, TC_WIDTH, CC_WIDTH);

    /* Linear 3 */
    dim3 dimGrid3(ceil(TC_WIDTH/warp_size), ceil(768/warp_size));
    TC_FP_Linear<<<dimGrid3,dimBlock>>>(Norm2_TC_output, Linear3_TC_weight_GPU, Linear3_TC_output, 
                                        Norm2_CC_output, Linear3_CC_weight_GPU, Linear3_CC_output, 
                                        768, TC_WIDTH, 768,
                                        768, CC_WIDTH, 768);

    /* Bias Add 3_1 */
    dim3 dimGrid3_1(ceil(768/block_size1), 1);
    dim3 dimblock3_1(block_size1, 1);
    TC_FP_Add<<<dimGrid3_1,dimblock3_1>>>(Linear3_TC_output, Linear3_TC_bias_GPU, Linear3_1_TC_output,
                                            Linear3_CC_output, Linear3_CC_bias_GPU, Linear3_1_CC_output,
                                            TC_WIDTH, CC_WIDTH);

    /* Layer Normalization 4 */
    dim3 dimGrid4(ceil(768/block_size1), ceil(TOTAL_WIDTH/block_size2));
    TC_FP_Normalization<<<dimGrid4,dimBlock>>>(Linear3_1_TC_output, Norm4_TC_output, Norm4_TC_gamma_GPU, Norm4_TC_beta_GPU, 
                                                Linear3_1_CC_output, Norm4_CC_output, Norm4_CC_gamma_GPU, Norm4_CC_beta_GPU,
                                                768, TC_WIDTH, CC_WIDTH);

    /* Dropout 5 */
    dim3 dimGrid5(ceil(768/block_size1), ceil(TOTAL_WIDTH/block_size2));
    TC_FP_Dropout<<<dimGrid5, dimBlock>>>(Norm4_TC_output, Drop5_TC_output, 
                                            Norm4_CC_output, Drop5_CC_output, 
                                            dropout_prob, TC_WIDTH, CC_WIDTH, 768, 768);
 
    /* Layer Normalization 6 */
    dim3 dimGrid6(ceil(768/block_size1), ceil(TOTAL_WIDTH/block_size2));
    TC_FP_Normalization<<<dimGrid6,dimBlock>>>(Drop5_TC_output, Norm6_TC_output, Norm6_TC_gamma_GPU, Norm6_TC_beta_GPU, 
                                                Drop5_CC_output, Norm6_CC_output, Norm6_CC_gamma_GPU, Norm6_CC_beta_GPU, 
                                                768, TC_WIDTH, CC_WIDTH);


    //////////////////////////////// First Layer ////////////////////////////////

    for(int i = 0; i < num_iteration; ++i){
        if(i == 0){
            /* Linear 7 */
            dim3 dimGrid7(ceil(TC_WIDTH/warp_size), ceil(2304/warp_size));
            TC_FP_Linear<<<dimGrid7,dimBlock>>>(Norm6_TC_output, Linear7_TC_weight_GPU[i], Linear7_TC_output[i], 
                                                Norm6_CC_output, Linear7_CC_weight_GPU[i], Linear7_CC_output[i], 
                                                2304, TC_WIDTH, 768,
                                                2304, CC_WIDTH, 768);
        }
        else{
            /* Linear 7 */
            dim3 dimGrid7(ceil(TC_WIDTH/warp_size), ceil(2304/warp_size));
            TC_FP_Linear<<<dimGrid7,dimBlock>>>(Norm20_TC_output[i-1], Linear7_TC_weight_GPU[i], Linear7_TC_output[i], 
                                                Norm20_CC_output[i-1], Linear7_CC_weight_GPU[i], Linear7_CC_output[i], 
                                                2304, TC_WIDTH, 768,
                                                2304, CC_WIDTH, 768);
        }

        /* Bias Add 7_1 */
        dim3 dimGrid7_1(ceil(2304/block_size1), 1);
        dim3 dimblock7_1(block_size1, 1);
        TC_FP_Add<<<dimGrid7_1,dimblock7_1>>>(Linear7_TC_output[i], Linear7_TC_bias_GPU[i], Linear7_1_TC_output[i],
                                            Linear7_CC_output[i], Linear7_CC_bias_GPU[i], Linear7_1_CC_output[i],
                                            TC_WIDTH, CC_WIDTH);

        /* Softmax 8 */
        dim3 dimGrid8(ceil(2304/block_size1), ceil(TOTAL_WIDTH/block_size2));
        TC_FP_Softmax<<<dimGrid8, dimBlock>>>(Linear7_1_TC_output[i], Soft8_TC_output[i],
                                                Linear7_1_CC_output[i], Soft8_CC_output[i], 
                                                2304, TC_WIDTH, CC_WIDTH);

        /* Dropout 9 */
        dim3 dimGrid9(ceil(768/block_size1), ceil(TOTAL_WIDTH/block_size2));
        TC_FP_Dropout<<<dimGrid9, dimBlock>>>(Soft8_TC_output[i], Drop9_TC_output[i], 
                                                Soft8_CC_output[i], Drop9_CC_output[i], 
                                                dropout_prob, TC_WIDTH, CC_WIDTH, 2304, 768);

        /* Linear 10 */
        dim3 dimGrid10(ceil(TC_WIDTH/warp_size), ceil(768/warp_size));
        TC_FP_Linear<<<dimGrid10,dimBlock>>>(Drop9_TC_output[i], Linear10_TC_weight_GPU[i], Linear10_TC_output[i],
                                            Drop9_CC_output[i], Linear10_CC_weight_GPU[i], Linear10_CC_output[i],
                                            768, TC_WIDTH, 768,
                                            768, CC_WIDTH, 768);

        /* Bias Add 10_1 */
        dim3 dimGrid10_1(ceil(768/block_size1), 1);
        dim3 dimblock10_1(block_size1, 1);
        TC_FP_Add<<<dimGrid10_1,dimblock10_1>>>(Linear10_TC_output[i], Linear10_TC_bias_GPU[i], Linear10_1_TC_output[i],
                                                Linear10_CC_output[i], Linear10_CC_bias_GPU[i], Linear10_1_CC_output[i],
                                                TC_WIDTH, CC_WIDTH);

        /* Dropout 11 */
        dim3 dimGrid11(ceil(768/block_size1), ceil(TOTAL_WIDTH/block_size2));
        TC_FP_Dropout<<<dimGrid11, dimBlock>>>(Linear10_1_TC_output[i], Drop11_TC_output[i], 
                                                Linear10_1_CC_output[i], Drop11_CC_output[i], 
                                                dropout_prob, TC_WIDTH, CC_WIDTH, 768, 768);

        /* Attention */

        /* Layer Normalization 13 */
        dim3 dimGrid13(ceil(768/block_size1), ceil(TOTAL_WIDTH/block_size2));
        TC_FP_Normalization<<<dimGrid13,dimBlock>>>(Drop11_TC_output[i], Norm13_TC_output[i], Norm13_TC_gamma_GPU[i], Norm13_TC_beta_GPU[i], 
                                                    Drop11_CC_output[i], Norm13_CC_output[i], Norm13_CC_gamma_GPU[i], Norm13_CC_beta_GPU[i], 
                                                    768, TC_WIDTH, CC_WIDTH);
        
        /* Linear 14 */
        dim3 dimGrid14(ceil(TC_WIDTH/warp_size), ceil(3072/warp_size));
        TC_FP_Linear<<<dimGrid14,dimBlock>>>(Norm13_TC_output[i], Linear14_TC_weight_GPU[i], Linear14_TC_output[i], 
                                            Norm13_CC_output[i], Linear14_CC_weight_GPU[i], Linear14_CC_output[i], 
                                            3072, TC_WIDTH, 768,
                                            3072, CC_WIDTH, 768);

        /* Bias Add 14_1 */
        dim3 dimGrid14_1(ceil(3072/block_size1), 1);
        dim3 dimblock14_1(block_size1, 1);
        TC_FP_Add<<<dimGrid14_1,dimblock14_1>>>(Linear14_TC_output[i], Linear14_TC_bias_GPU[i], Linear14_1_TC_output[i],
                                                Linear14_CC_output[i], Linear14_CC_bias_GPU[i], Linear14_1_CC_output[i],
                                                TC_WIDTH, CC_WIDTH);

        /* Gelu 15 */
        dim3 dimGrid15(ceil(3072/block_size1), ceil(TOTAL_WIDTH/block_size2));
        TC_FP_Gelu<<<dimGrid15,dimBlock>>>(Linear14_1_TC_output[i], Gelu15_TC_output[i], 
                                            Linear14_1_CC_output[i], Gelu15_CC_output[i],
                                            TC_WIDTH, CC_WIDTH, 3072);

        /* Dropout 16 */
        dim3 dimGrid16(ceil(3072/block_size1), ceil(TOTAL_WIDTH/block_size2));
        TC_FP_Dropout<<<dimGrid16, dimBlock>>>(Gelu15_TC_output[i], Drop16_TC_output[i], 
                                                Gelu15_CC_output[i], Drop16_CC_output[i], 
                                                dropout_prob, TC_WIDTH, CC_WIDTH, 3072, 3072);

        /* Linear 17 */
        dim3 dimGrid17(ceil(TC_WIDTH/warp_size), ceil(768/warp_size));
        TC_FP_Linear<<<dimGrid17,dimBlock>>>(Drop16_TC_output[i], Linear17_TC_weight_GPU[i], Linear17_TC_output[i], 
                                            Drop16_CC_output[i], Linear17_CC_weight_GPU[i], Linear17_CC_output[i],
                                            768, TC_WIDTH, 3072,
                                            768, CC_WIDTH, 3072);
        
        /* Bias Add 17_1 */
        dim3 dimGrid17_1(ceil(768/block_size1), 1);
        dim3 dimblock17_1(block_size1, 1);
        TC_FP_Add<<<dimGrid17_1,dimblock17_1>>>(Linear17_TC_output[i], Linear17_TC_bias_GPU[i], Linear17_1_TC_output[i],
                                                Linear17_CC_output[i], Linear17_CC_bias_GPU[i], Linear17_1_CC_output[i],
                                                TC_WIDTH, CC_WIDTH);

        /* Dropout 18 */
        dim3 dimGrid18(ceil(768/block_size1), ceil(TOTAL_WIDTH/block_size2));
        TC_FP_Dropout<<<dimGrid18, dimBlock>>>(Linear17_1_TC_output[i], Drop18_TC_output[i], 
                                                Linear17_1_CC_output[i], Drop18_CC_output[i], 
                                                dropout_prob, TC_WIDTH, CC_WIDTH, 768, 768);

        /* Feedforward */

        /* Layer Normalization 20 */
        dim3 dimGrid20(ceil(768/block_size1), ceil(TOTAL_WIDTH/block_size2));
        TC_FP_Normalization<<<dimGrid20,dimBlock>>>(Drop18_TC_output[i], Norm20_TC_output[i], Norm20_TC_gamma_GPU[i], Norm20_TC_beta_GPU[i], 
                                                    Drop18_CC_output[i], Norm20_CC_output[i], Norm20_CC_gamma_GPU[i], Norm20_CC_beta_GPU[i], 
                                                    768, TC_WIDTH, CC_WIDTH);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Inference Time: %fms\n", milliseconds);

    // Freeing CPU Memory
    delete[] Rearrange_TC_input;
    delete[] Rearrange_CC_input;

    delete[] Norm2_TC_gamma_CPU;
    delete[] Norm2_CC_gamma_CPU;
    delete[] Norm2_TC_beta_CPU;
    delete[] Norm2_CC_beta_CPU;

    delete[] Linear3_TC_weight_CPU;
    delete[] Linear3_TC_bias_CPU;
    delete[] Linear3_CC_weight_CPU;
    delete[] Linear3_CC_bias_CPU;

    delete[] Norm4_TC_gamma_CPU;
    delete[] Norm4_TC_beta_CPU;
    delete[] Norm4_CC_gamma_CPU;
    delete[] Norm4_CC_beta_CPU;

    delete[] Norm6_TC_gamma_CPU;
    delete[] Norm6_TC_beta_CPU;
    delete[] Norm6_CC_gamma_CPU;
    delete[] Norm6_CC_beta_CPU;

    for(int i = 0; i < num_iteration; ++i) {
        delete[] Linear7_TC_weight_CPU[i];
        delete[] Linear7_TC_bias_CPU[i];
        delete[] Linear7_CC_weight_CPU[i];
        delete[] Linear7_CC_bias_CPU[i];

        delete[] Linear10_TC_weight_CPU[i];
        delete[] Linear10_TC_bias_CPU[i];
        delete[] Linear10_CC_weight_CPU[i];
        delete[] Linear10_CC_bias_CPU[i];

        delete[] Norm13_TC_gamma_CPU[i];
        delete[] Norm13_TC_beta_CPU[i];
        delete[] Norm13_CC_gamma_CPU[i];
        delete[] Norm13_CC_beta_CPU[i];

        delete[] Linear14_TC_weight_CPU[i];
        delete[] Linear14_TC_bias_CPU[i];
        delete[] Linear14_CC_weight_CPU[i];
        delete[] Linear14_CC_bias_CPU[i];

        delete[] Linear17_TC_weight_CPU[i];
        delete[] Linear17_TC_bias_CPU[i];
        delete[] Linear17_CC_weight_CPU[i];
        delete[] Linear17_CC_bias_CPU[i];

        delete[] Norm20_TC_gamma_CPU[i];
        delete[] Norm20_TC_beta_CPU[i];
        delete[] Norm20_CC_gamma_CPU[i];
        delete[] Norm20_CC_beta_CPU[i];
    }

    // Freeing GPU Memory
    hipFree(Rearrange1_TC_output_GPU);
    hipFree(Rearrange1_CC_output_GPU);

    hipFree(Norm2_TC_gamma_GPU);
    hipFree(Norm2_CC_gamma_GPU);
    hipFree(Norm2_TC_beta_GPU);
    hipFree(Norm2_CC_beta_GPU);
    hipFree(Norm2_TC_output);
    hipFree(Norm2_CC_output);

    hipFree(Linear3_TC_weight_GPU);
    hipFree(Linear3_TC_bias_GPU);
    hipFree(Linear3_TC_output);
    hipFree(Linear3_1_TC_output);
    hipFree(Linear3_CC_weight_GPU);
    hipFree(Linear3_CC_bias_GPU);
    hipFree(Linear3_CC_output);
    hipFree(Linear3_1_CC_output);

    hipFree(Norm4_TC_gamma_GPU);
    hipFree(Norm4_TC_beta_GPU);
    hipFree(Norm4_TC_output);
    hipFree(Norm4_CC_gamma_GPU);
    hipFree(Norm4_CC_beta_GPU);
    hipFree(Norm4_CC_output);

    hipFree(Drop5_TC_output);
    hipFree(Drop5_CC_output);

    hipFree(Norm6_TC_gamma_GPU);
    hipFree(Norm6_TC_beta_GPU);
    hipFree(Norm6_TC_output);
    hipFree(Norm6_CC_gamma_GPU);
    hipFree(Norm6_CC_beta_GPU);
    hipFree(Norm6_CC_output);

    for(int i = 0; i < num_iteration; ++i) {
        hipFree(Linear7_TC_weight_GPU[i]);
        hipFree(Linear7_TC_bias_GPU[i]);
        hipFree(Linear7_TC_output[i]);
        hipFree(Linear7_1_TC_output[i]);
        hipFree(Linear7_CC_weight_GPU[i]);
        hipFree(Linear7_CC_bias_GPU[i]);
        hipFree(Linear7_CC_output[i]);
        hipFree(Linear7_1_CC_output[i]);

        hipFree(Soft8_TC_output[i]);
        hipFree(Soft8_CC_output[i]);

        hipFree(Drop9_TC_output[i]);
        hipFree(Drop9_CC_output[i]);

        hipFree(Linear10_TC_weight_GPU[i]);
        hipFree(Linear10_TC_bias_GPU[i]);
        hipFree(Linear10_TC_output[i]);
        hipFree(Linear10_1_TC_output[i]);
        hipFree(Linear10_CC_weight_GPU[i]);
        hipFree(Linear10_CC_bias_GPU[i]);
        hipFree(Linear10_CC_output[i]);
        hipFree(Linear10_1_CC_output[i]);

        hipFree(Drop11_TC_output[i]);
        hipFree(Drop11_CC_output[i]);

        hipFree(Norm13_TC_gamma_GPU[i]);
        hipFree(Norm13_TC_beta_GPU[i]);
        hipFree(Norm13_TC_output[i]);
        hipFree(Norm13_CC_gamma_GPU[i]);
        hipFree(Norm13_CC_beta_GPU[i]);
        hipFree(Norm13_CC_output[i]);

        hipFree(Linear14_TC_weight_GPU[i]);
        hipFree(Linear14_TC_bias_GPU[i]);
        hipFree(Linear14_TC_output[i]);
        hipFree(Linear14_1_TC_output[i]);
        hipFree(Linear14_CC_weight_GPU[i]);
        hipFree(Linear14_CC_bias_GPU[i]);
        hipFree(Linear14_CC_output[i]);
        hipFree(Linear14_1_CC_output[i]);

        hipFree(Gelu15_TC_output[i]);
        hipFree(Gelu15_CC_output[i]);

        hipFree(Drop16_TC_output[i]);
        hipFree(Drop16_CC_output[i]);

        hipFree(Linear17_TC_weight_GPU[i]);
        hipFree(Linear17_TC_bias_GPU[i]);
        hipFree(Linear17_TC_output[i]);
        hipFree(Linear17_1_TC_output[i]);
        hipFree(Linear17_CC_weight_GPU[i]);
        hipFree(Linear17_CC_bias_GPU[i]);
        hipFree(Linear17_CC_output[i]);
        hipFree(Linear17_1_CC_output[i]);

        hipFree(Drop18_TC_output[i]);
        hipFree(Drop18_CC_output[i]);

        hipFree(Norm20_TC_gamma_GPU[i]);
        hipFree(Norm20_TC_beta_GPU[i]);
        hipFree(Norm20_TC_output[i]);
        hipFree(Norm20_CC_gamma_GPU[i]);
        hipFree(Norm20_CC_beta_GPU[i]);
        hipFree(Norm20_CC_output[i]);
    }

    return 0;
}
