#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <mma.h>

#include "Only_TC_Functions.cuh"

#define warp_size 32
#define block_size1 32
#define block_size2 24

#define WIDTH 784

void initializeRandom_int8(int8_t* array, int size) {
    for (int i = 0; i < size; ++i) {
        array[i] = rand() % 11;
    }
}

void initializeRandom_float(float* array, int size) {
    for (int i = 0; i < size; ++i) {
        array[i] = static_cast<float>(rand()) / RAND_MAX; // 0부터 1까지의 랜덤값
    }
}

void rearrange(int8_t *input, int8_t *output, int output_rows, int output_cols) {
    for (int row = 0; row < output_rows; ++row) {
        for (int col = 0; col < output_cols; ++col) {
            int idx = row * output_cols + col;
            if (row < output_rows && col < output_cols) {
                int out_row = idx / output_cols;
                int out_col = idx % output_cols;
                int in_idx = (out_row * 14 + out_col / 48) * 224 + out_col % 48;

                output[idx] = input[in_idx];
            }
        }
    }
}

int main(){

    ///// Initalizing Input Data, Weight, Bias, Gamma, Beta /////
    /* Rearange 1 */
    int8_t *Rearrange_input = new int8_t[224*224*3];
    initializeRandom_int8(Rearrange_input, 224*224*3);

    /* Measuring Preprocessng Time*/
    // clock_t Pre_Processing_Start, Pre_Processing_End;
    // double Pre_Processing_Time;

    // Pre_Processing_Start = clock();

    int8_t *Rearrange1_output_GPU;
    hipMalloc(&Rearrange1_output_GPU, (WIDTH*192) * sizeof(int8_t));
    hipMemcpy(Rearrange1_output_GPU, Rearrange_input, (WIDTH*192) * sizeof(int8_t), hipMemcpyHostToDevice);

    /* Layer Normalization 2 */
    int8_t *Norm2_gamma_CPU = new int8_t[192];
    initializeRandom_int8(Norm2_gamma_CPU, 192);
    int8_t *Norm2_gamma_GPU;
    hipMalloc(&Norm2_gamma_GPU, 192 * sizeof(int8_t));
    hipMemcpy(Norm2_gamma_GPU, Norm2_gamma_CPU, 192 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Norm2_beta_CPU = new int8_t[192];
    initializeRandom_int8(Norm2_beta_CPU, 192);
    int8_t *Norm2_beta_GPU;
    hipMalloc(&Norm2_beta_GPU, 192 * sizeof(int8_t));
    hipMemcpy(Norm2_beta_GPU, Norm2_beta_CPU, 192 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Norm2_output;
    hipMalloc(&Norm2_output, (WIDTH*192) * sizeof(int8_t));

    /* Linear 3 */
    int8_t *Linear3_weight_CPU = new int8_t[192 * 768];
    initializeRandom_int8(Linear3_weight_CPU, 192 * 768);
    int8_t *Linear3_weight_GPU;
    hipMalloc(&Linear3_weight_GPU, (192 * 768) * sizeof(int8_t));
    hipMemcpy(Linear3_weight_GPU, Linear3_weight_CPU, (192 * 768) * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Linear3_bias_CPU = new int8_t[768];
    initializeRandom_int8(Linear3_bias_CPU, 768);
    int8_t *Linear3_bias_GPU;
    hipMalloc(&Linear3_bias_GPU, 768 * sizeof(int8_t));
    hipMemcpy(Linear3_bias_GPU, Linear3_bias_CPU, 768 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Linear3_output;
    hipMalloc(&Linear3_output, (WIDTH*768) * sizeof(int8_t));
    int8_t *Linear3_1_output;
    hipMalloc(&Linear3_1_output, (WIDTH*768) * sizeof(int8_t));

    /* Layer Normalization 4 */
    int8_t *Norm4_gamma_CPU = new int8_t[768];
    initializeRandom_int8(Norm4_gamma_CPU, 768);
    int8_t *Norm4_gamma_GPU;
    hipMalloc(&Norm4_gamma_GPU, 768 * sizeof(int8_t));
    hipMemcpy(Norm4_gamma_GPU, Norm4_gamma_CPU, 768 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Norm4_beta_CPU = new int8_t[768];
    initializeRandom_int8(Norm4_beta_CPU, 768);
    int8_t *Norm4_beta_GPU;
    hipMalloc(&Norm4_beta_GPU, 768 * sizeof(int8_t));
    hipMemcpy(Norm4_beta_GPU, Norm4_beta_CPU, 768 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Norm4_output;
    hipMalloc(&Norm4_output, (WIDTH*768) * sizeof(int8_t));

    /* Dropout 5 */
    int8_t *Drop5_output;
    hipMalloc(&Drop5_output, (WIDTH*768) * sizeof(int8_t));

    const float dropout_prob = 0.5f;

    /* Layer Normalization 6 */
    int8_t *Norm6_gamma_CPU = new int8_t[768];
    initializeRandom_int8(Norm6_gamma_CPU, 768);
    int8_t *Norm6_gamma_GPU;
    hipMalloc(&Norm6_gamma_GPU, 768 * sizeof(int8_t));
    hipMemcpy(Norm6_gamma_GPU, Norm6_gamma_CPU, 768 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Norm6_beta_CPU = new int8_t[768];
    initializeRandom_int8(Norm6_beta_CPU, 768);
    int8_t *Norm6_beta_GPU;
    hipMalloc(&Norm6_beta_GPU, 768 * sizeof(int8_t));
    hipMemcpy(Norm6_beta_GPU, Norm6_beta_CPU, 768 * sizeof(int8_t), hipMemcpyHostToDevice);

    int8_t *Norm6_output;
    hipMalloc(&Norm6_output, (WIDTH*768) * sizeof(int8_t));

    //// Iteration Start
    int num_iteration = 12;

    /* Linear 7 */
    int8_t *Linear7_weight_CPU[num_iteration];
    int8_t *Linear7_weight_GPU[num_iteration];
    int8_t *Linear7_bias_CPU[num_iteration];
    int8_t *Linear7_bias_GPU[num_iteration];
    int8_t *Linear7_output[num_iteration];
    int8_t *Linear7_1_output[num_iteration];

    /* Softmax 8 */
    int8_t *Soft8_output[num_iteration];

    /* Dropout 9 */
    int8_t *Drop9_output[num_iteration];

    /* Linear 10 */
    int8_t *Linear10_weight_CPU[num_iteration];
    int8_t *Linear10_weight_GPU[num_iteration];
    int8_t *Linear10_bias_CPU[num_iteration];
    int8_t *Linear10_bias_GPU[num_iteration];
    int8_t *Linear10_output[num_iteration];
    int8_t *Linear10_1_output[num_iteration];

    /* Dropout 11 */
    int8_t *Drop11_output[num_iteration];

    /* Layer Normalization 13 */
    int8_t *Norm13_gamma_CPU[num_iteration];
    int8_t *Norm13_gamma_GPU[num_iteration];
    int8_t *Norm13_beta_CPU[num_iteration];
    int8_t *Norm13_beta_GPU[num_iteration];
    int8_t *Norm13_output[num_iteration];

    /* Linear 14 */
    int8_t *Linear14_weight_CPU[num_iteration];
    int8_t *Linear14_weight_GPU[num_iteration];
    int8_t *Linear14_bias_CPU[num_iteration];
    int8_t *Linear14_bias_GPU[num_iteration];
    int8_t *Linear14_output[num_iteration];
    int8_t *Linear14_1_output[num_iteration];

    /* Gelu 15 */
    int8_t *Gelu15_output[num_iteration];

    /* Dropout 16 */
    int8_t *Drop16_output[num_iteration];

    /* Linear 17 */
    int8_t *Linear17_weight_CPU[num_iteration];
    int8_t *Linear17_weight_GPU[num_iteration];
    int8_t *Linear17_bias_CPU[num_iteration];
    int8_t *Linear17_bias_GPU[num_iteration];
    int8_t *Linear17_output[num_iteration];
    int8_t *Linear17_1_output[num_iteration];

    /* Dropout 18 */
    int8_t *Drop18_output[num_iteration];

    /* Layer Normalization 20 */
    int8_t *Norm20_gamma_CPU[num_iteration];
    int8_t *Norm20_gamma_GPU[num_iteration];
    int8_t *Norm20_beta_CPU[num_iteration];
    int8_t *Norm20_beta_GPU[num_iteration];
    int8_t *Norm20_output[num_iteration];
    
    for(int i = 0; i < num_iteration; ++i){
        /* Linear 7 */
        Linear7_weight_CPU[i] = new int8_t[768 * 2304];
        initializeRandom_int8(Linear7_weight_CPU[i], 768 * 2304);
        
        hipMalloc(&Linear7_weight_GPU[i], (768 * 2304) * sizeof(int8_t));
        hipMemcpy(Linear7_weight_GPU[i], Linear7_weight_CPU[i], (768 * 2304) * sizeof(int8_t), hipMemcpyHostToDevice);

        Linear7_bias_CPU[i] = new int8_t[2304];
        initializeRandom_int8(Linear7_bias_CPU[i], 2304);
        
        hipMalloc(&Linear7_bias_GPU[i], 2304 * sizeof(int8_t));
        hipMemcpy(Linear7_bias_GPU[i], Linear7_bias_CPU[i], 2304 * sizeof(int8_t), hipMemcpyHostToDevice);

        
        hipMalloc(&Linear7_output[i], (WIDTH*2304) * sizeof(int8_t));
        hipMalloc(&Linear7_1_output[i], (WIDTH*2304) * sizeof(int8_t));

        /* Softmax 8 */
        hipMalloc(&Soft8_output[i], (WIDTH*2304) * sizeof(int8_t));

        /* Dropout 9 */
        hipMalloc(&Drop9_output[i], (WIDTH*768) * sizeof(int8_t));

        /* Linear 10 */
        Linear10_weight_CPU[i] = new int8_t[768 * 768];
        initializeRandom_int8(Linear10_weight_CPU[i], 768 * 768);
       
        hipMalloc(&Linear10_weight_GPU[i], (768 * 768) * sizeof(int8_t));
        hipMemcpy(Linear10_weight_GPU[i], Linear10_weight_CPU[i], (768 * 768) * sizeof(int8_t), hipMemcpyHostToDevice);

        Linear10_bias_CPU[i] = new int8_t[768];
        initializeRandom_int8(Linear10_bias_CPU[i], 768);
        
        hipMalloc(&Linear10_bias_GPU[i], 768 * sizeof(int8_t));
        hipMemcpy(Linear10_bias_GPU[i], Linear10_bias_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);

        hipMalloc(&Linear10_output[i], (WIDTH*768) * sizeof(int8_t));
        hipMalloc(&Linear10_1_output[i], (WIDTH*768) * sizeof(int8_t));

        /* Dropout 11 */
        hipMalloc(&Drop11_output[i], (WIDTH*768) * sizeof(int8_t));

        /* Layer Normalization 13 */
        Norm13_gamma_CPU[i] = new int8_t[768];
        initializeRandom_int8(Norm13_gamma_CPU[i], 768);

        hipMalloc(&Norm13_gamma_GPU[i], 768 * sizeof(int8_t));
        hipMemcpy(Norm13_gamma_GPU[i], Norm13_gamma_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);

        Norm13_beta_CPU[i] = new int8_t[768];
        initializeRandom_int8(Norm13_beta_CPU[i], 768);
        
        hipMalloc(&Norm13_beta_GPU[i], 768 * sizeof(int8_t));
        hipMemcpy(Norm13_beta_GPU[i], Norm13_beta_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);

        hipMalloc(&Norm13_output[i], (WIDTH*768) * sizeof(int8_t));

        /* Linear 14 */
        Linear14_weight_CPU[i] = new int8_t[768 * 3072];
        initializeRandom_int8(Linear14_weight_CPU[i], 768 * 3072);

        hipMalloc(&Linear14_weight_GPU[i], (768 * 3072) * sizeof(int8_t));
        hipMemcpy(Linear14_weight_GPU[i], Linear14_weight_CPU[i], (768 * 3072) * sizeof(int8_t), hipMemcpyHostToDevice);

        Linear14_bias_CPU[i] = new int8_t[3072];
        initializeRandom_int8(Linear14_bias_CPU[i], 3072);

        hipMalloc(&Linear14_bias_GPU[i], 3072 * sizeof(int8_t));
        hipMemcpy(Linear14_bias_GPU[i], Linear14_bias_CPU[i], 3072 * sizeof(int8_t), hipMemcpyHostToDevice);

        hipMalloc(&Linear14_output[i], (WIDTH*3072) * sizeof(int8_t));
        hipMalloc(&Linear14_1_output[i], (WIDTH*3072) * sizeof(int8_t));

        /* Gelu 15 */
        hipMalloc(&Gelu15_output[i], (WIDTH*3072) * sizeof(int8_t));

        /* Dropout 16 */;
        hipMalloc(&Drop16_output[i], (WIDTH*3072) * sizeof(int8_t));

        /* Linear 17 */
        Linear17_weight_CPU[i] = new int8_t[768 * 3072];
        initializeRandom_int8(Linear17_weight_CPU[i], 768 * 3072);

        hipMalloc(&Linear17_weight_GPU[i], (768 * 3072) * sizeof(int8_t));
        hipMemcpy(Linear17_weight_GPU[i], Linear17_weight_CPU[i], (768 * 3072) * sizeof(int8_t), hipMemcpyHostToDevice);

        Linear17_bias_CPU[i] = new int8_t[768];
        initializeRandom_int8(Linear17_bias_CPU[i], 768);

        hipMalloc(&Linear17_bias_GPU[i], 3072 * sizeof(int8_t));
        hipMemcpy(Linear17_bias_GPU[i], Linear17_bias_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);

        hipMalloc(&Linear17_output[i], (WIDTH*768) * sizeof(int8_t));
        hipMalloc(&Linear17_1_output[i], (WIDTH*768) * sizeof(int8_t));

        /* Dropout 18 */
        hipMalloc(&Drop18_output[i], (WIDTH*768) * sizeof(int8_t));

        /* Layer Normalization 20 */
        Norm20_gamma_CPU[i] = new int8_t[768];
        initializeRandom_int8(Norm20_gamma_CPU[i], 768);

        hipMalloc(&Norm20_gamma_GPU[i], 768 * sizeof(int8_t));
        hipMemcpy(Norm20_gamma_GPU[i], Norm20_gamma_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);

        Norm20_beta_CPU[i] = new int8_t[768];
        initializeRandom_int8(Norm20_beta_CPU[i], 768);

        hipMalloc(&Norm20_beta_GPU[i], 768 * sizeof(int8_t));
        hipMemcpy(Norm20_beta_GPU[i], Norm20_beta_CPU[i], 768 * sizeof(int8_t), hipMemcpyHostToDevice);

        hipMalloc(&Norm20_output[i], (WIDTH*768) * sizeof(int8_t));
    }

    // Pre_Processing_End = clock();

    // Pre_Processing_Time = ((double) (Pre_Processing_End - Pre_Processing_Start)) / CLOCKS_PER_SEC;

    // printf("Pre_Processing_Time: %f s\n", Pre_Processing_Time);

    ///// Starting Computation /////

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    dim3 dimBlock(block_size1, block_size2);

    /* Layer Normalization 2 */
    dim3 dimGrid2(ceil(192/block_size1), ceil(WIDTH/block_size2));
    Only_TC_Normalization<<<dimGrid2,dimBlock>>>(Rearrange1_output_GPU, Norm2_output, Norm2_gamma_GPU, Norm2_beta_GPU, 768, WIDTH);

    /* Linear 3 */
    dim3 dimGrid3(ceil(WIDTH/warp_size), ceil(768/warp_size));
    Only_TC_Linear<<<dimGrid3,dimBlock>>>(Norm2_output, Linear3_weight_GPU, Linear3_output, 768, WIDTH, 768);

    /* Bias Add 3_1 */
    dim3 dimGrid3_1(ceil(768/block_size1), 1);
    dim3 dimblock3_1(block_size1, 1);
    Only_TC_Add<<<dimGrid3_1,dimblock3_1>>>(Linear3_output, Linear3_bias_GPU, Linear3_1_output, WIDTH);

    /* Layer Normalization 4 */
    dim3 dimGrid4(ceil(768/block_size1), ceil(WIDTH/block_size2));
    Only_TC_Normalization<<<dimGrid4,dimBlock>>>(Linear3_1_output, Norm4_output, Norm4_gamma_GPU, Norm4_beta_GPU, 768, WIDTH);

    /* Dropout 5 */
    dim3 dimGrid5(ceil(768/block_size1), ceil(WIDTH/block_size2));
    Only_TC_Dropout<<<dimGrid5, dimBlock>>>(Norm4_output, Drop5_output, dropout_prob, WIDTH, 768, 768);
 
    /* Layer Normalization 6 */
    dim3 dimGrid6(ceil(768/block_size1), ceil(WIDTH/block_size2));
    Only_TC_Normalization<<<dimGrid6,dimBlock>>>(Drop5_output, Norm6_output, Norm6_gamma_GPU, Norm6_beta_GPU, 768, WIDTH);


    //////////////////////////////// First Layer ////////////////////////////////

    for(int i = 0; i < num_iteration; ++i){
        if(i == 0){
            /* Linear 7 */
            dim3 dimGrid7(ceil(WIDTH/warp_size), ceil(2304/warp_size));
            Only_TC_Linear<<<dimGrid7,dimBlock>>>(Norm6_output, Linear7_weight_GPU[i], Linear7_output[i], 2304, WIDTH, 768);
        }
        else{
            /* Linear 7 */
            dim3 dimGrid7(ceil(WIDTH/warp_size), ceil(2304/warp_size));
            Only_TC_Linear<<<dimGrid7,dimBlock>>>(Norm20_output[i-1], Linear7_weight_GPU[i], Linear7_output[i], 2304, WIDTH, 768);
        }

        /* Bias Add 7_1 */
        dim3 dimGrid7_1(ceil(2304/block_size1), 1);
        dim3 dimblock7_1(block_size1, 1);
        Only_TC_Add<<<dimGrid7_1,dimblock7_1>>>(Linear7_output[i], Linear7_bias_GPU[i], Linear7_1_output[i],WIDTH);

        /* Softmax 8 */
        dim3 dimGrid8(ceil(2304/block_size1), ceil(WIDTH/block_size2));
        Only_TC_Softmax<<<dimGrid8, dimBlock>>>(Linear7_1_output[i], Soft8_output[i], WIDTH, 2304);

        /* Dropout 9 */
        dim3 dimGrid9(ceil(768/block_size1), ceil(WIDTH/block_size2));
        Only_TC_Dropout<<<dimGrid9, dimBlock>>>(Soft8_output[i], Drop9_output[i], dropout_prob, WIDTH, 2304, 768);

        /* Linear 10 */
        dim3 dimGrid10(ceil(WIDTH/warp_size), ceil(768/warp_size));
        Only_TC_Linear<<<dimGrid10,dimBlock>>>(Drop9_output[i], Linear10_weight_GPU[i], Linear10_output[i], 768, WIDTH, 768);

        /* Bias Add 10_1 */
        dim3 dimGrid10_1(ceil(768/block_size1), 1);
        dim3 dimblock10_1(block_size1, 1);
        Only_TC_Add<<<dimGrid10_1,dimblock10_1>>>(Linear10_output[i], Linear10_bias_GPU[i], Linear10_1_output[i], WIDTH);

        /* Dropout 11 */
        dim3 dimGrid11(ceil(768/block_size1), ceil(WIDTH/block_size2));
        Only_TC_Dropout<<<dimGrid11, dimBlock>>>(Linear10_1_output[i], Drop11_output[i], dropout_prob, WIDTH, 768, 768);

        /* Attention */

        /* Layer Normalization 13 */
        dim3 dimGrid13(ceil(768/block_size1), ceil(WIDTH/block_size2));
        Only_TC_Normalization<<<dimGrid13,dimBlock>>>(Drop11_output[i], Norm13_output[i], Norm13_gamma_GPU[i], Norm13_beta_GPU[i], 768, WIDTH);


        /* Linear 14 */
        dim3 dimGrid14(ceil(WIDTH/warp_size), (3072/warp_size));
        Only_TC_Linear<<<dimGrid14,dimBlock>>>(Norm13_output[i], Linear14_weight_GPU[i], Linear14_output[i], 3072, WIDTH, 768);

        /* Bias Add 14_1 */
        dim3 dimGrid14_1(ceil(3072/block_size1), 1);
        dim3 dimblock14_1(block_size1, 1);
        Only_TC_Add<<<dimGrid14_1,dimblock14_1>>>(Linear14_output[i], Linear14_bias_GPU[i], Linear14_1_output[i], WIDTH);

        /* Gelu 15 */
        dim3 dimGrid15(ceil(3072/block_size1), ceil(WIDTH/block_size2));
        Only_TC_Gelu<<<dimGrid15,dimBlock>>>(Linear14_1_output[i], Gelu15_output[i], WIDTH);

        /* Dropout 16 */
        dim3 dimGrid16(ceil(3072/block_size1), ceil(WIDTH/block_size2));
        Only_TC_Dropout<<<dimGrid16, dimBlock>>>(Gelu15_output[i], Drop16_output[i], dropout_prob, WIDTH, 3072, 3072);

        /* Linear 17 */
        dim3 dimGrid17(ceil(WIDTH/warp_size), ceil(768/warp_size));
        Only_TC_Linear<<<dimGrid17,dimBlock>>>(Drop16_output[i], Linear17_weight_GPU[i], Linear17_output[i], 768, WIDTH, 3072);

        /* Bias Add 17_1 */
        dim3 dimGrid17_1(ceil(768/block_size1), 1);
        dim3 dimblock17_1(block_size1, 1);
        Only_TC_Add<<<dimGrid17_1,dimblock17_1>>>(Linear17_output[i], Linear17_bias_GPU[i], Linear17_1_output[i], WIDTH);

        /* Dropout 18 */
        dim3 dimGrid18(ceil(768/block_size1), ceil(WIDTH/block_size2));
        Only_TC_Dropout<<<dimGrid18, dimBlock>>>(Linear17_1_output[i], Drop18_output[i], dropout_prob, WIDTH, 768, 768);

        /* Feedforward */

        /* Layer Normalization 20 */
        dim3 dimGrid20(ceil(768/block_size1), ceil(WIDTH/block_size2));
        Only_TC_Normalization<<<dimGrid20,dimBlock>>>(Drop18_output[i], Norm20_output[i], Norm20_gamma_GPU[i], Norm20_beta_GPU[i], 768, WIDTH);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Inference Time: %fms\n", milliseconds);

    // Freeing CPU Memory
    delete[] Rearrange_input;

    delete[] Norm2_gamma_CPU;
    delete[] Norm2_beta_CPU;
    delete[] Linear3_weight_CPU;
    delete[] Linear3_bias_CPU;
    delete[] Norm4_gamma_CPU;
    delete[] Norm4_beta_CPU;
    delete[] Norm6_gamma_CPU;
    delete[] Norm6_beta_CPU;

    for(int i = 0; i < num_iteration; ++i) {
        delete[] Linear7_weight_CPU[i];
        delete[] Linear7_bias_CPU[i];
        delete[] Linear10_weight_CPU[i];
        delete[] Linear10_bias_CPU[i];
        delete[] Norm13_gamma_CPU[i];
        delete[] Norm13_beta_CPU[i];
        delete[] Linear14_weight_CPU[i];
        delete[] Linear14_bias_CPU[i];
        delete[] Linear17_weight_CPU[i];
        delete[] Linear17_bias_CPU[i];
        delete[] Norm20_gamma_CPU[i];
        delete[] Norm20_beta_CPU[i];
    }

    // Freeing GPU Memory
    hipFree(Rearrange1_output_GPU);

    hipFree(Norm2_gamma_GPU);
    hipFree(Norm2_beta_GPU);
    hipFree(Norm2_output);

    hipFree(Linear3_weight_GPU);
    hipFree(Linear3_bias_GPU);
    hipFree(Linear3_output);
    hipFree(Linear3_1_output);

    hipFree(Norm4_gamma_GPU);
    hipFree(Norm4_beta_GPU);
    hipFree(Norm4_output);

    hipFree(Drop5_output);

    hipFree(Norm6_gamma_GPU);
    hipFree(Norm6_beta_GPU);
    hipFree(Norm6_output);

    for(int i = 0; i < num_iteration; ++i) {
        hipFree(Linear7_weight_GPU[i]);
        hipFree(Linear7_bias_GPU[i]);
        hipFree(Linear7_output[i]);
        hipFree(Linear7_1_output[i]);

        hipFree(Soft8_output[i]);

        hipFree(Drop9_output[i]);

        hipFree(Linear10_weight_GPU[i]);
        hipFree(Linear10_bias_GPU[i]);
        hipFree(Linear10_output[i]);
        hipFree(Linear10_1_output[i]);

        hipFree(Drop11_output[i]);

        hipFree(Norm13_gamma_GPU[i]);
        hipFree(Norm13_beta_GPU[i]);
        hipFree(Norm13_output[i]);

        hipFree(Linear14_weight_GPU[i]);
        hipFree(Linear14_bias_GPU[i]);
        hipFree(Linear14_output[i]);
        hipFree(Linear14_1_output[i]);

        hipFree(Gelu15_output[i]);

        hipFree(Drop16_output[i]);

        hipFree(Linear17_weight_GPU[i]);
        hipFree(Linear17_bias_GPU[i]);
        hipFree(Linear17_output[i]);
        hipFree(Linear17_1_output[i]);

        hipFree(Drop18_output[i]);

        hipFree(Norm20_gamma_GPU[i]);
        hipFree(Norm20_beta_GPU[i]);
        hipFree(Norm20_output[i]);
    }

    return 0;
}
