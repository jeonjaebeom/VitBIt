
#include <hip/hip_runtime.h>
__global__ void TC_INT_FP_Add(const int8_t* TC_input, const int8_t* TC_weight, int8_t* TC_output,
                            const int *CC_input_int, const int *CC_weight_int, int *CC_output_int,
                            const float *CC_input_fp, const float *CC_weight_fp, float *CC_output_fp,
                            int TC_width, int CC_width_half) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    for(int i = 0; i < TC_width; i++)
        TC_output[idx + i] = TC_input[idx + i] + TC_weight[idx];
    for(int i = 0; i < CC_width_half; i++){
        CC_output_int[idx + i] = CC_input_int[idx + i] + CC_weight_int[idx];
        CC_output_fp[idx + i] = CC_input_fp[idx + i] + CC_weight_fp[idx];
    }
}
